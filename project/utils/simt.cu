#include "hip/hip_runtime.h"
extern "C" {

#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

#include "simt.h"

#ifndef CUDA_BLOCK_SIZE
#define CUDA_BLOCK_SIZE 16  // TODO change this value to the best you found during your analysis
#endif

/*
    GPU kernel performing a matrix multiplication
    __global__ specifies that it will execute on the device (= the GPU)
 */
__global__ void kernel_multiply_matrix(uint32_t *A, uint32_t *B, uint32_t *C, uint32_t K) {

    // TODO write the kernel code
    // Here you can use blockIdx, blockDim and threadIdx
    uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < K && col < K) {
        // TODO we want a multiplication!
        C[row * K + col] = A[row * K + col] + B[row * K + col];
        
        #error SIMT Not implemented
    }
}


/*
    Helper function that allocates GPU memory, copies the data to the GPU, amd launches the kernel 
    It will execute on the host (= the CPU)
    Feel free to modify it!
*/
void multiply_matrix_simt(uint32_t *matrix1, uint32_t *matrix2, uint32_t *result, uint32_t K) {
    /* Allocate GPU memory for the matrices */
    uint32_t *A_d, *B_d, *C_d;
    hipMalloc(&A_d, K * K * sizeof(uint32_t));
    hipMalloc(&B_d, K * K * sizeof(uint32_t));
    hipMalloc(&C_d, K * K * sizeof(uint32_t));

    /* Copy matrices A and B from host to device */
    hipMemcpy(A_d, matrix1, K * K * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(B_d, matrix2, K * K * sizeof(uint32_t), hipMemcpyHostToDevice);

    /* Define the block and grid dimensions */
    // TODO what block size do you want?
    dim3 threadsPerBlock(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE); // e.g., 16x16 threads per block
    dim3 numBlocks((K + threadsPerBlock.x - 1) / threadsPerBlock.x, (K + threadsPerBlock.y - 1) / threadsPerBlock.y);   // try to understand why such computations!
    
    /* Launch the kernel */
    kernel_multiply_matrix <<< numBlocks, threadsPerBlock >>> (A_d, B_d, C_d, K);

    /* Wait for the kernel to finish */
    hipDeviceSynchronize();

    /* Copy the result matrix C from device to host */
    hipMemcpy(result, C_d, K * K * sizeof(int), hipMemcpyDeviceToHost);

    /* Free device memory */
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

} /* extern "C" */